
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(int *data);

int main(){

  int *d_data, h_data = 0;
  hipMalloc((void **)&d_data, sizeof(int));
  hipMemcpy(d_data, &h_data, sizeof(int), hipMemcpyHostToDevice);
  mykernel<<<1,1>>>(d_data);
  hipMemcpy(&h_data, d_data, sizeof(int), hipMemcpyDeviceToHost);
  printf("data = %d\n", h_data);
  return 0;
}
